#include "hip/hip_runtime.h"
#include "utils.h"
#include <thrust/scan.h>


template <typename scalar_t>
__global__ void composite_alpha_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    const scalar_t T_threshold,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> alphas,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // front to back compositing
    int samples = 0; scalar_t T = 1.0f;

    while (samples < N_samples) {
        const int s = start_idx + samples;
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        const scalar_t w = a * T;

        alphas[s] = a;
        ws[s] = w;
        T *= 1.0f-a;

        if (T <= T_threshold) break; // ray has enough opacity
        samples++;
    }
}


std::vector<torch::Tensor> composite_alpha_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor deltas,
    const torch::Tensor rays_a,
    const float T_threshold
){
    const int N = sigmas.size(0), N_rays = rays_a.size(0);

    auto alphas = torch::zeros({N}, sigmas.options());
    auto ws = torch::zeros({N}, sigmas.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_alpha_fw_cu", 
    ([&] {
            composite_alpha_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            T_threshold,
            alphas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {alphas, ws};
}

template <typename scalar_t>
__global__ void composite_train_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgbs,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> normals_pred,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sems,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    const scalar_t T_threshold,
    const int64_t classes, 
    torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> total_samples,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> opacity,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgb,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> normal_pred,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sem,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= opacity.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // front to back compositing
    int samples = 0; scalar_t T = 1.0f;

    while (samples < N_samples) {
        const int s = start_idx + samples;
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        const scalar_t w = a * T;

        rgb[ray_idx][0] += w*rgbs[s][0];
        rgb[ray_idx][1] += w*rgbs[s][1];
        rgb[ray_idx][2] += w*rgbs[s][2];
        normal_pred[ray_idx][0] += w*normals_pred[s][0];
        normal_pred[ray_idx][1] += w*normals_pred[s][1];
        normal_pred[ray_idx][2] += w*normals_pred[s][2];
        depth[ray_idx] += w*ts[s];
        for (int i=0;i<classes;i++) {
            sem[ray_idx][i] += w*sems[s][i];
        }
        opacity[ray_idx] += w;
        ws[s] = w;
        T *= 1.0f-a;

        if (T <= T_threshold) break; // ray has enough opacity
        samples++;
    }
    total_samples[ray_idx] = samples;
}


std::vector<torch::Tensor> composite_train_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor rgbs,
    const torch::Tensor normals_pred,
    const torch::Tensor sems,
    const torch::Tensor clips,
    const torch::Tensor dinos,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const float T_threshold,
    const int classes
){
    const int N = sigmas.size(0), N_rays = rays_a.size(0);

    auto opacity = torch::zeros({N_rays}, sigmas.options());
    auto depth = torch::zeros({N_rays}, sigmas.options());
    auto rgb = torch::zeros({N_rays, 3}, sigmas.options());
    auto normal_pred = torch::zeros({N_rays, 3}, sigmas.options());\
    auto sem = torch::zeros({N_rays, classes}, sigmas.options());
    auto ws = torch::zeros({N}, sigmas.options());
    auto total_samples = torch::zeros({N_rays}, torch::dtype(torch::kLong).device(sigmas.device()));

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_train_fw_cu", 
    ([&] {
        composite_train_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgbs.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            normals_pred.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            sems.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            T_threshold,
            classes,
            total_samples.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
            opacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgb.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            normal_pred.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            sem.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {total_samples, opacity, depth, rgb, normal_pred, sem, ws};
}


template <typename scalar_t>
__global__ void composite_train_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dopacity,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_ddepth,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_drgb,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_dnormal_pred,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_dsem,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dws,
    scalar_t* __restrict__ dL_dws_times_ws,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgbs,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> normals_pred,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> opacity,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgb,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> normal_pred,
    const scalar_t T_threshold,
    const int64_t classes,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dsigmas,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_drgbs,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_dnormals_pred,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_dsems
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= opacity.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // front to back compositing
    int samples = 0;
    scalar_t R = rgb[ray_idx][0], G = rgb[ray_idx][1], B = rgb[ray_idx][2];
    scalar_t O = opacity[ray_idx], D = depth[ray_idx];
    scalar_t T = 1.0f, r = 0.0f, g = 0.0f, b = 0.0f, d = 0.0f;

    // compute prefix sum of dL_dws * ws
    // [a0, a1, a2, a3, ...] -> [a0, a0+a1, a0+a1+a2, a0+a1+a2+a3, ...]
    thrust::inclusive_scan(thrust::device,
                           dL_dws_times_ws+start_idx,
                           dL_dws_times_ws+start_idx+N_samples,
                           dL_dws_times_ws+start_idx);
    scalar_t dL_dws_times_ws_sum = dL_dws_times_ws[start_idx+N_samples-1];

    while (samples < N_samples) {
        const int s = start_idx + samples;
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        const scalar_t w = a * T;

        r += w*rgbs[s][0]; g += w*rgbs[s][1]; b += w*rgbs[s][2];
        d += w*ts[s];
        T *= 1.0f-a;

        // compute gradients by math...
        dL_drgbs[s][0] = dL_drgb[ray_idx][0]*w;
        dL_drgbs[s][1] = dL_drgb[ray_idx][1]*w;
        dL_drgbs[s][2] = dL_drgb[ray_idx][2]*w;
        // compute gradients by math...
        dL_dnormals_pred[s][0] = dL_dnormal_pred[ray_idx][0]*w;
        dL_dnormals_pred[s][1] = dL_dnormal_pred[ray_idx][1]*w;
        dL_dnormals_pred[s][2] = dL_dnormal_pred[ray_idx][2]*w;

        for (int i=0;i<classes;i++){
            dL_dsems[s][i] = dL_dsem[ray_idx][i]*w;
        }

        dL_dsigmas[s] = deltas[s] * (
            dL_drgb[ray_idx][0]*(rgbs[s][0]*T-(R-r)) + 
            dL_drgb[ray_idx][1]*(rgbs[s][1]*T-(G-g)) + 
            dL_drgb[ray_idx][2]*(rgbs[s][2]*T-(B-b)) + 
            dL_dopacity[ray_idx]*(1-O) + 
            dL_ddepth[ray_idx]*(ts[s]*T-(D-d)) + 
            T*dL_dws[s]-(dL_dws_times_ws_sum-dL_dws_times_ws[s])
        );

        if (T <= T_threshold) break; // ray has enough opacity
        samples++;
    }
}


std::vector<torch::Tensor> composite_train_bw_cu(
    const torch::Tensor dL_dopacity,
    const torch::Tensor dL_ddepth,
    const torch::Tensor dL_drgb,
    const torch::Tensor dL_dnormal_pred,
    const torch::Tensor dL_dsem,
    const torch::Tensor dL_dws,
    const torch::Tensor sigmas,
    const torch::Tensor rgbs,
    const torch::Tensor normals_pred,
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const torch::Tensor opacity,
    const torch::Tensor depth,
    const torch::Tensor rgb,
    const torch::Tensor normal_pred,
    const float T_threshold,
    const int classes
){
    const int N = sigmas.size(0), N_rays = rays_a.size(0);

    auto dL_dsigmas = torch::zeros({N}, sigmas.options());
    auto dL_drgbs = torch::zeros({N, 3}, sigmas.options());
    auto dL_dnormals_pred = torch::zeros({N, 3}, sigmas.options());
    auto dL_dsems = torch::zeros({N, classes}, sigmas.options());

    auto dL_dws_times_ws = dL_dws * ws; // auxiliary input

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_train_bw_cu", 
    ([&] {
        composite_train_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dopacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_ddepth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_drgb.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            dL_dnormal_pred.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            dL_dsem.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            dL_dws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_dws_times_ws.data_ptr<scalar_t>(),
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgbs.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            normals_pred.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            opacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgb.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            normal_pred.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            T_threshold,
            classes,
            dL_dsigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_drgbs.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            dL_dnormals_pred.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            dL_dsems.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {dL_dsigmas, dL_drgbs, dL_dnormals_pred, dL_dsems};
}


template <typename scalar_t>
__global__ void composite_test_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> rgbs,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> normals,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> normals_raw,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> sems,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> hits_t,
    torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> alive_indices,
    const scalar_t T_threshold,
    const int64_t classes,
    const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> N_eff_samples,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> opacity,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgb,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> normal,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> normal_raw,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sem
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= alive_indices.size(0)) return;

    if (N_eff_samples[n]==0){ // no hit
        alive_indices[n] = -1;
        return;
    }

    const size_t r = alive_indices[n]; // ray index

    // front to back compositing
    int s = 0; scalar_t T = 1-opacity[r];

    while (s < N_eff_samples[n]) {
        const scalar_t a = 1.0f - __expf(-sigmas[n][s]*deltas[n][s]);
        const scalar_t w = a * T;

        rgb[r][0] += w*rgbs[n][s][0];
        rgb[r][1] += w*rgbs[n][s][1];
        rgb[r][2] += w*rgbs[n][s][2];
        depth[r] += w*ts[n][s];
        opacity[r] += w;
        normal[r][0] += w*normals[n][s][0];
        normal[r][1] += w*normals[n][s][1];
        normal[r][2] += w*normals[n][s][2];
        normal_raw[r][0] += w*normals_raw[n][s][0];
        normal_raw[r][1] += w*normals_raw[n][s][1];
        normal_raw[r][2] += w*normals_raw[n][s][2];
        for(int i=0;i<classes;i++){
            sem[r][i] += w*sems[n][s][i];
        }
        T *= 1.0f-a;

        if (T <= T_threshold){ // ray has enough opacity
            alive_indices[n] = -1;
            break;
        }
        s++;
    }
}

void composite_test_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor rgbs,
    const torch::Tensor normals,
    const torch::Tensor normals_raw,
    const torch::Tensor sems,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor hits_t,
    torch::Tensor alive_indices,
    const float T_threshold,
    const int classes,
    const torch::Tensor N_eff_samples,
    torch::Tensor opacity,
    torch::Tensor depth,
    torch::Tensor rgb,
    torch::Tensor normal,
    torch::Tensor normal_raw,
    torch::Tensor sem
){
    const int N_rays = alive_indices.size(0);

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_test_fw_cu", 
    ([&] {
        composite_test_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            rgbs.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            normals.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            normals_raw.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            sems.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            hits_t.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            alive_indices.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
            T_threshold,
            classes,
            N_eff_samples.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
            opacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgb.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            normal.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            normal_raw.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            sem.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));
}
